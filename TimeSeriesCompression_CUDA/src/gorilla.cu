#include "hip/hip_runtime.h"
#include "compressors.h"
#include "utils/encode_utils.h"
#include "utils/cuda_common_utils.h"
#include "utils/bit_writer.cuh"

__constant__ uint32_t c_count; // the number of data points
__constant__ uint64_t* c_uncompressed_t; // uncompressed timestamps on device
__constant__ uint64_t* c_uncompressed_v; // uncompressed values on device
__constant__ uint32_t* c_offs; // offset of uncompressed data divided by lane
__constant__ byte* c_compressed_t; // compressed timestamps on device
__constant__ byte* c_compressed_v; // compressed values on device
__constant__ uint32_t* c_len_t; // length of compressed timestamps on device
__constant__ uint32_t* c_len_v; // length of compressed values on device
__constant__ uint64_t* c_decompressed_t; // decompressed timestamps on device
__constant__ uint64_t* c_decompressed_v; // decompressed values on device

// Timestamps compression kernal 
__global__ static void timestamp_compress_kernal();

// Compress timestamps in specific scope of uncompressed timestamps
__device__ static inline void timestamp_compress_device(
    int start, int end, BitWriter *bitWriter, uint32_t thdIdx
);

// Compress timestamps on GPU
void timestamp_compress_gorilla_gpu(
    ByteBuffer* tsByteBuffer,
    uint32_t blocks,
    uint32_t warps
) {
    // divide the uncompressed data into frames according to the 
    // total number of threads
    uint32_t
        count, // the number of data points
        thdOfBlock, // the number of threads within per block
        thd, // the total number of needed threads
        *offs, // start offsets of data that each thread will compress
        frame, // the length of data that each thread will compress
        padding = 0, // unused pos in the used last frame
        left = 0; // unused threads in the last block

    count = tsByteBuffer->length / BYTES_OF_LONG_LONG;
    thdOfBlock = WARPSIZE*warps;
    thd = thdOfBlock*blocks;
    frame = (count + thd - 1) / thd;

    // if count <= MIN_FRAME_SIZE, use just one thread to compress
    if (count <= MIN_FRAME_SIZE) {
        frame = count; blocks = 1; thdOfBlock = 1; thd = 1;
    }
    else if (frame < MIN_FRAME_SIZE) {// else if frame is too small
        frame = MIN_FRAME_SIZE;
        // recalculatre the number of needed threads
        thd = (count + frame - 1) / frame;
        padding = frame - count % frame;
        if (thd < MAX_THREADS_PER_BLOCK) {
            // use just one block to compress
            blocks = 1;
            //warps = (thd + WARPSIZE - 1) / WARPSIZE;
            thdOfBlock = thd;
        }
        else {
            // use block as less as possible to compress according to the 
            // number of threads within per block
            blocks = (thd + thdOfBlock - 1) / thdOfBlock;
            left = blocks*thdOfBlock - thd;
            thd = blocks*thdOfBlock;
        }
    }

    // construct the offsets array, each scope of frame is [offs[i],offs[i+1])
    offs = (uint32_t*)malloc(BYTES_OF_INT*(thd + 1));
    for (int i = 0; i <= thd - left - 1; i++)
        offs[i] = i*frame;
    offs[thd - left] = (thd - left)*frame - padding;
    for (int i = thd - left + 1; i <= thd; i++)
        offs[i] = offs[thd - left];

    // allocate device memory and tranport data to GPU
    uint64_t* d_uncompressed_t; // uncompressed timestamps
    byte* d_compressed_t; // compressed timestamps
    uint32_t* d_len_t; // length of compressed timestamps on device
    uint32_t* d_offs; // data offset of threads
    checkCudaError(hipMalloc((void**)&d_uncompressed_t, tsByteBuffer->length));
    checkCudaError(hipMalloc((void**)&d_compressed_t, tsByteBuffer->length));
    checkCudaError(hipMalloc((void**)&d_len_t, BYTES_OF_INT*thd));
    checkCudaError(hipMalloc((void**)&d_offs, BYTES_OF_INT*(thd + 1)));
    checkCudaError(hipMemcpy(
        d_uncompressed_t, tsByteBuffer->buffer,
        tsByteBuffer->length, hipMemcpyHostToDevice
    ));
    checkCudaError(hipMemcpy(
        d_offs, offs,
        BYTES_OF_INT*(thd + 1), hipMemcpyHostToDevice
    ));
    checkCudaError(hipMemcpyToSymbol(HIP_SYMBOL(c_uncompressed_t), &d_uncompressed_t, sizeof(void *)));
    checkCudaError(hipMemcpyToSymbol(HIP_SYMBOL(c_compressed_t), &d_compressed_t, sizeof(void *)));
    checkCudaError(hipMemcpyToSymbol(HIP_SYMBOL(c_len_t), &d_len_t, sizeof(void *)));
    checkCudaError(hipMemcpyToSymbol(HIP_SYMBOL(c_offs), &d_offs, sizeof(void *)));
    checkCudaError(hipMemcpyToSymbol(HIP_SYMBOL(c_count), &count, sizeof(uint32_t)));

    // initiate kernal
    timestamp_compress_kernal <<<blocks, thdOfBlock >>> ();

    checkCudaError(hipDeviceSynchronize());

    // allocate cpu memory for compressed data, and copy data from GPU to CPU
    byte* compressed_t = (byte*)malloc(tsByteBuffer->length);
    uint32_t* len_t = (uint32_t*)malloc(BYTES_OF_INT*thd);
    checkCudaError(hipMemcpy(
        compressed_t, d_compressed_t, tsByteBuffer->length,
        hipMemcpyDeviceToHost)
    );
    checkCudaError(hipMemcpy(
        len_t, d_len_t, BYTES_OF_INT*thd,
        hipMemcpyDeviceToHost)
    );

    // Print the compressed data(just for test)
    printCompressedData(compressed_t, offs, len_t, thd);

    // return compressed data and it's length
}


// Timestamps compression kernal
__global__ static void timestamp_compress_kernal() {
    // declare
    uint32_t
        thdIdx, // thread index within grid
        start, // start offset of uncompressed data in current thread
        end; // end offset of uncompressed data in current thread

    thdIdx = threadIdx.x + blockIdx.x*blockDim.x;
    start = c_offs[thdIdx];
    end = c_offs[thdIdx + 1];

    // construct
    ByteBuffer byteBuffer;
    byteBuffer.buffer = (byte*)(((uint64_t*)c_compressed_t)+start); // start pos for compression
    byteBuffer.length = 0;
    BitWriter bitWriter;
    bitWriter.byteBuffer = &byteBuffer;
    bitWriter.cacheByte = 0;
    bitWriter.leftBits = BITS_OF_BYTE;

    // compress the timestamps within this thread
    timestamp_compress_device(start, end, &bitWriter, thdIdx);

}

// Compress timestamps in specific scope of uncompressed timestamps
__device__ static inline void timestamp_compress_device(
    int start, int end, BitWriter *bitWriter, uint32_t thdIdx
) {
    // declaration
    int64_t timestamp, prevTimestamp;
    int32_t newDelta, deltaOfDelta, prevDelta;
    uint32_t leastBitLength;
    uint64_t *tsBuffer = c_uncompressed_t;

    if (start == 0) {// If current timestamp is the first one
        prevTimestamp = 0;
        prevDelta = 0;
    }
    else {// else 'start'>=32
        prevTimestamp = tsBuffer[start - 1];
        prevDelta = prevTimestamp - tsBuffer[start - 2];
    }

    // compress every timestamp in the scope into the compressed buffer
    for (int cur = start; cur < end; cur++) {
        // Calculate the delta of delta of timestamp.
        timestamp = tsBuffer[cur];

        // PS: since original implementation in gorilla paper requires that delta-of-delta
        // of timestamps can be stored by a signed 32-bit value, it doesn't support
        // compression timestamps in millisecond as good as second.
        newDelta = (int32_t)(timestamp - prevTimestamp);
        deltaOfDelta = newDelta - prevDelta;

        // if current delta and previous delta is same
        if (deltaOfDelta == 0) {
            // Write '0' bit as control bit(i.e. previous and current delta value is same).
            bitWriterWriteZeroBit(bitWriter);
        }
        else {
            // Tips: since deltaOfDelta == 0 is unoccupied, we can utilize it to cover a larger range.
            if (deltaOfDelta > 0) deltaOfDelta--;
            // convert signed value to unsigned value for compression.
            deltaOfDelta = encodeZigZag32(deltaOfDelta);

            //leastBitLength = BITS_OF_INT - leadingZerosCount32(deltaOfDelta);
            leastBitLength = BITS_OF_INT - __clz(deltaOfDelta);
            // match the deltaOfDelta to the these case as follow.
            switch (leastBitLength) {
            case 0:
            case 1:
            case 2:
            case 3:
            case 4:
            case 5:
            case 6:
            case 7:
                // '10'+7
                bitWriterWriteBits(bitWriter, 0b10, 2);
                bitWriterWriteBits(bitWriter, deltaOfDelta, 7);
                break;
            case 8:
            case 9:
                // '110'+9
                bitWriterWriteBits(bitWriter, 0b110, 3);
                bitWriterWriteBits(bitWriter, deltaOfDelta, 9);
                break;
            case 10:
            case 11:
            case 12:
                // '1110'+12
                bitWriterWriteBits(bitWriter, 0b1110, 4);
                bitWriterWriteBits(bitWriter, deltaOfDelta, 12);
                break;
            default:
                // '1111'+32
                // Write '1111' control bits.
                bitWriterWriteBits(bitWriter, 0b1111, 4);
                // Since it only takes 4 bytes(i.e. 32 bits) to save a unix timestamp input second, we write
                // delta-of-delta using 32 bits.
                bitWriterWriteBits(bitWriter, deltaOfDelta, 32);
                break;
            }

            // update previous delta of timestamp
            prevDelta = newDelta;
        }
        // update previous timestamp
        prevTimestamp = timestamp;
    }

    // write the left bits in cached byte into the buffer.
    bitWriterFlush(bitWriter);

    // write the byte length of compressed timestamps
    c_len_t[thdIdx] = bitWriter->byteBuffer->length;
}