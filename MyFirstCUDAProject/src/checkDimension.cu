#include "hip/hip_runtime.h"
#include <stdio.h>

#include "hip/hip_runtime.h"
#include ""

__global__ void checkIndex(void) {
    printf(
        "gridDim:(%d,%d,%d), blockDim:(%d,%d,%d), threadIdx:(%d,%d,%d), blockIdx:(%d,%d,%d).\n",
        gridDim.x, gridDim.y, gridDim.z, blockDim.x, blockDim.y, blockDim.z,
        threadIdx.x, threadIdx.y, threadIdx.z, blockIdx.x, blockIdx.y, blockIdx.z
    );
}

int main(int argc, char** argv) {
    // Define the number of elements;
    const int numOfElements = 32;

    // Define grid and block structures
    dim3 block(3);
    dim3 grid((numOfElements + block.x - 1) / block.x);

    // Check grid and block dimension on host side.
    printf("grid.x:%d, grid.y:%d, grid.z:%d.\n", grid.x, grid.y, grid.z);
    printf("block.x:%d, block.y:%d, block.z:%d.\n", block.x, block.y, block.z);

    // Check grid and block dimension on device side.
    checkIndex << <grid, block >> > ();

    // Reset device memory allocation and status.
    hipDeviceReset();

    return 0;
}