#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <stdio.h>

// Define a kernal function by "__global__" keyword.
__global__ void helloFromGPU() {
    printf("Hello World! This is CPU(Host) thread %d .\n", threadIdx.x);
}

int main(void) {
    using namespace std;
    cout << "Hello world from CPU!\n" << endl;
    // Say "Hello World" from CPU(Host).
    printf("Hello world from CPU!\n");

    // Define a cuda status variable.
    hipError_t cudaStatus;

    // Launch a kernal function on GPU(Device).
    helloFromGPU << <1, 10 >> > ();

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addVectorKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        return 1;
    }

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipError_t::hipSuccess)
    {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addVectorKernel!\n", cudaStatus);
        return 1;
    }

    return 0;
}